#include "hip/hip_runtime.h"
/*I have taken the help of lzhengchun/matrix-cuda github repository to get this code working.*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16

__global__ void g_mat_mul(int *a,int *b, int *c, int m, int n, int k)
{ 
    int r = blockIdx.y * blockDim.y + threadIdx.y; 
    int cl = blockIdx.x * blockDim.x + threadIdx.x;
    int temp = 0;
    if( cl < k && r < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            temp += a[r * n + i] * b[i * k + cl];
        }
        c[r * k + cl] = temp;
    }
} 

int main(int argc, char const *argv[])
{
    int m_rows, n_col_row, k_col;
   
    printf("please type in A=mxn B=nxk \n");
    scanf("%d %d %d", &m_rows, &n_col_row, &k_col);
    int *h_a_cpu, *h_b_cpu, *h_c_cpu, *h_cc_cpu;
    hipHostMalloc((void **) &h_a_cpu, sizeof(int)*m_rows*n_col_row);
    hipHostMalloc((void **) &h_b_cpu, sizeof(int)*n_col_row*k_col);
    hipHostMalloc((void **) &h_c_cpu, sizeof(int)*m_rows*k_col);
    hipHostMalloc((void **) &h_cc_cpu, sizeof(int)*m_rows*k_col);

 for (int i = 0; i < m_rows; ++i) {
        for (int j = 0; j < n_col_row; ++j) {
            h_a_cpu[i * n_col_row + j] = rand() % 1024;
        }
}

for (int i = 0; i < n_col_row; ++i) {
        for (int j = 0; j < k_col; ++j) {
            h_b_cpu[i * k_col + j] = rand() % 1024;
        }
}

float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

hipEvent_t start, stop;

hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start, 0);

 int *d_a_gpu, *d_b_gpu, *d_c_gpu;
    hipMalloc((void **) &d_a_gpu, sizeof(int)*m_rows*n_col_row);
    hipMalloc((void **) &d_b_gpu, sizeof(int)*n_col_row*k_col);
    hipMalloc((void **) &d_c_gpu, sizeof(int)*m_rows*k_col);
    hipMemcpy(d_a_gpu, h_a_cpu, sizeof(int)*m_rows*n_col_row, hipMemcpyHostToDevice);
    hipMemcpy(d_b_gpu, h_b_cpu, sizeof(int)*n_col_row*k_col, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m_rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k_col + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);


 g_mat_mul<<<dimGrid, dimBlock>>>(d_a_gpu, d_b_gpu, d_c_gpu, m_rows, n_col_row, k_col);

hipMemcpy(h_a_cpu, d_c_gpu, sizeof(int)*m_rows*k_col, hipMemcpyDeviceToHost);

hipDeviceSynchronize();

hipEventRecord(stop, 0);
hipEventSynchronize(stop);

hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m_rows, n_col_row, n_col_row, k_col, gpu_elapsed_time_ms);

hipFree(d_a_gpu);
    hipFree(d_b_gpu);
    hipFree(d_c_gpu);
    hipHostFree(h_a_cpu);
    hipHostFree(h_b_cpu);
    hipHostFree(h_c_cpu);
    hipHostFree(h_cc_cpu);
    return 0;
}
