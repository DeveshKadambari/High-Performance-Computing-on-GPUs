#include "hip/hip_runtime.h"
/*
College: University of Massachusetts Lowell
EECE 7110:High-Performance Comp. on GPUs
Semester: Spring 2018
Student : 01639617
Project : Assignment_3
Professor : Dr.Hang Liu
Due date: 4/16/2018
Authors : Sai Sri Devesh Kadambari
*/


#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
using namespace std;

#define zero 0

__global__ void gpu_up_swing(int *a,int *c, int m)
{
  __shared__ int smem[1];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int p=16384,n=1,pwr;					//p=m/blockdim*griddim
    while((tid<(p*n))&&(tid<(32000000)))
{
    __syncthreads();                                                    //wait until all the threads in the block reach this point
for(int depth =0;depth< __logf ( 16384) ;depth++)
{	pwr=__powf(depth, 2);
	pwr_1=__powf(depth, 2+1);
if(tid%pwr_1==0){
   a[tid+pwr_1-1]+=a[tid+pwr_1-1];
}
__syncthreads();
}
	gpu_down_swing(*a,*c,m);
     smem[1]=a[p*n];
	tid+=p;
	a[tid]+=smem[1];
	n++;
	__syncthreads(); 
}
}

__global__ void gpu_down_swing(int *a,int *c, int m)
{
  __shared__ int smem[1];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int p=16384,n=1,pwr;					//p=m/blockdim*griddim
    while((tid<(p*n))&&(tid<(32000000)))
{
    __syncthreads();                                                    //wait until all the threads in the block reach this point
for(int depth =0;depth< __logf ( 16384) ;depth++)
{	pwr=__powf(depth, 2);
	pwr_1=__powf(depth, 2+1);

   a[tid+pwr_1-1]+=a[tid+pwr_1-1];
__syncthreads();
}
     smem[1]=a[p*n];
	tid+=p;
	a[tid]+=smem[1];
	n++;
	__syncthreads(); 
}
}


int main(int argc, char const *argv[])
{ int m;
    printf("please type in m(size)  \n");
    scanf("%d", &m);

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_c;// *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m);
    hipHostMalloc((void **) &h_c, sizeof(int)*m);

 for (int i = 0; i < m; ++i) {
       
  h_a[i] = rand() % 1024;          //loading random values
       
}

clock_t t;
    t = clock();
int *d_a,*d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m);
    hipMalloc((void **) &d_c, sizeof(int)*m);
    hipMemcpy(d_a, h_a, sizeof(int)*m, hipMemcpyHostToDevice);
    dim3 dimGrid(128);
    dim3 dimBlock(128);


 gpu_up_swing<<<dimGrid, dimBlock>>>(d_a,d_c, m);

hipMemcpy(h_c, d_c, sizeof(int)*m, hipMemcpyDeviceToHost);
gpu_up_swing<<<dimGrid, dimBlock>>>(d_a,d_c, m);
hipDeviceSynchronize();

t = clock()-t;
 double time_taken = ((double)t)/CLOCKS_PER_SEC;
printf("Time elapsed on operation of %d: %lf ms.\n\n", m,(time_taken/1000));

hipFree(d_a);
    
    hipFree(d_c);
    hipHostFree(h_a);
   
    hipHostFree(h_c);
    //hipHostFree(h_cc);
    return 0;
}

