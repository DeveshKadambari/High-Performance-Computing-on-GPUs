#include "hip/hip_runtime.h"
/*
College: University of Massachusetts Lowell
EECE 7110:High-Performance Comp. on GPUs
Semester: Spring 2018
Student : 01639617
Project : Assignment_2
Professor : Dr.Hang Liu
Due date: 2/12/2017
Authors : Sai Sri Devesh Kadambari
*/


#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
using namespace std;

#define zero 0

__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{
  __shared__ int smem[250];
    int row = blockIdx.x ;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = m/gridDim.x;                                             //step=80
    int index_begin = row * step;
    int index_end= (row+ 1) * step;       //Block 0= 0->80
    float f=(blockDim.x)/2;int k_b;
    __syncthreads();                                                    //wait until all the threads in the block reach this point
    for(int i=(index_begin);i<index_end;i++)                            //Row=0->80
        { smem[tid] =a[i * (blockDim.x)+tid] *b[tid];                   //save multiplication value into the smem buffer
         __syncthreads();                                               //wait until all the threads reach this point
                for(int j=((blockDim.x)/2);j>0;j=ceilf(f))            //i=250/2 is 125->62.5(63)->(63-1)->(21)->20->(10)->(5)->(3)->(2)->(1)
                { k_b=2*f;
                if(((k_b)%2!=0) && (threadIdx.x == (j-1)))
                	{
                        smem[threadIdx.x -1]+=smem[threadIdx.x];
                        j=j-1;
                        f=j;
               		}

                        if(threadIdx.x < j)
                        { int temp =smem[threadIdx.x]+smem[threadIdx.x + j];
                         smem[threadIdx.x]=temp;
                        }
			 __syncthreads();
                        f=f/2;
                }
          c[i]=smem[zero];
        }

}

int main(int argc, char const *argv[])
{
    int m, n, k;
    printf("please type in m=A_rows n=A_columns and k=B_columns \n");
    scanf("%d %d %d", &m, &n, &k);

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c;// *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k);
    //hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);

 for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
  h_a[i * n + j] = rand() % 1024;          //h_a[row_variable*Max_column + column_variable]
        }
}

for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
}

float gpu_elapsed_time_ms;

hipEvent_t start, stop;

hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start, 0);
clock_t t;
    t = clock();
int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);
    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    //unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    //unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(128);
    dim3 dimBlock(256);


 gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);

hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);

hipDeviceSynchronize();

hipEventRecord(stop, 0);
hipEventSynchronize(stop);
t = clock()-t;
 double time_taken = ((double)t)/CLOCKS_PER_SEC;
hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %lf ms.\n\n", m, n, n, k, (time_taken/1000));

hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    //hipHostFree(h_cc);
    return 0;
}

